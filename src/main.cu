#include <SDL2/SDL.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include "../include/Game.h"

int main(int argc, char** argv)
{
    srand(time(NULL));

    Game * g = NULL;
    g = initGame(1024, 768);
    if(g == NULL)
    {
        return -1;
    }

    playGame(g);
    
    deleteGame(g);
    
    return 0;
}
