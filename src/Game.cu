#include "hip/hip_runtime.h"
#include "../include/Game.h"

Game* initGame(unsigned int width, unsigned int height)
{
    if (SDL_Init(SDL_INIT_VIDEO) != 0 )
    {
        fprintf(stderr,"SDL_init error :  %s\n",SDL_GetError());
        return NULL;
    }

    Game* ptr     = (Game*)malloc(sizeof(Game)); 
    ptr->m_width  = width;
    ptr->m_height = height;
    ptr->m_window = NULL;
    ptr->m_window = SDL_CreateWindow("rule110Cuda",
            SDL_WINDOWPOS_UNDEFINED,
            SDL_WINDOWPOS_UNDEFINED,
            ptr->m_width,
            ptr->m_height,
            SDL_WINDOW_OPENGL);

    if(ptr->m_window == NULL)
    {
        fprintf(stderr, "SDL_CreateWindow error : %s\n", SDL_GetError());
        free(ptr);
        ptr = NULL;
    }

    ptr->m_renderer = NULL;
    ptr->m_renderer = SDL_CreateRenderer(ptr->m_window, 0, SDL_RENDERER_ACCELERATED);

    ptr->m_cellDataSize = sizeof(char) * ptr->m_width * ptr->m_height / (CELL_SIZE);

    unsigned int tmpWidth = ptr->m_width / (CELL_SIZE);
    unsigned int tmpHeight = ptr->m_height / (CELL_SIZE);

    hipMalloc((void**)&(ptr->m_width_device), sizeof(unsigned int));
    hipMalloc((void**)&(ptr->m_height_device), sizeof(unsigned int));

    hipMemcpy(ptr->m_width_device, &tmpWidth, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(ptr->m_height_device, &tmpHeight, sizeof(unsigned int), hipMemcpyHostToDevice);

    ptr->m_cellData_host = (char*) malloc(ptr->m_cellDataSize);
    hipMalloc((void**) &(ptr->m_cellData_device), ptr->m_cellDataSize);

    resetGame(ptr, 1);

    return ptr;
}

void playGame(Game* ptr)
{
    SDL_Event event;
    short run = 0;

    const unsigned int sizeX = ptr->m_width / (CELL_SIZE);
    const unsigned int sizeY = ptr->m_height / (CELL_SIZE); 

    while(run == 0)
    {
        while (SDL_PollEvent(&event))
        {
            switch (event.type)
            {
                case SDL_QUIT:
                    {
                        run = 1;
                        break;
                    }
                case SDL_KEYUP:
                    {
                        if(event.key.keysym.sym == SDLK_r){
                            resetGame(ptr, 0);
                        }else if(event.key.keysym.sym == SDLK_e){
                            resetGame(ptr, 1);
                        }
                        break;
                    }
            }
        }
        SDL_SetRenderDrawColor(ptr->m_renderer, 0, 0, 0, 255);
        SDL_RenderClear(ptr->m_renderer);

        unsigned int x = 0;
        unsigned int y = 0;

        for(y=0; y < sizeY; ++y){
            for(x=0; x < sizeX; ++x){
                SDL_Rect r;
                r.x = x * CELL_SIZE;
                r.y = y * CELL_SIZE;
                r.w = CELL_SIZE;
                r.h = CELL_SIZE;


                if(ptr->m_cellData_host[x + y * sizeX] == 1){
                    SDL_SetRenderDrawColor(ptr->m_renderer, 255, 255, 255, 255);
                } else {
                    SDL_SetRenderDrawColor(ptr->m_renderer, 0, 0, 0, 255);
                }
                SDL_RenderFillRect(ptr->m_renderer, &r);
            }
        }

        SDL_RenderPresent(ptr->m_renderer);
        SDL_Delay(16);
    }
}

__device__ void applyRule(char* left, char* middle, char* right, char* res){
    char a = *left;
    char b = *middle;
    char c = *right;

    if(a == 0 && b == 0 && c == 0){
        *res = 0;
    }else if(a == 0 && b == 0 && c == 1){
        *res = 1;
    }else if(a == 0 && b == 1 && c == 0){
        *res = 1;
    }else if(a == 0 && b == 1 && c == 1){
        *res = 1;
    }else if(a == 1 && b == 0 && c == 0){
        *res = 0;
    }else if(a == 1 && b == 0 && c == 1){
        *res = 1;
    }else if(a == 1 && b == 1 && c == 0){
        *res = 1;
    }else if(a == 1 && b == 1 && c == 1){
        *res = 0;
    }
}

__global__ void computeCell(char* cellData, unsigned int* width, unsigned int* height)
{
    int y = 0;
    int x = 0;

    /*
       printf("width : %d\n", *width);
       printf("height : %d\n", *height);
       printf("dimblock : %d\n", blockDim.x);
       printf("threadid : %d\n", threadIdx.x);
     */

    for(y=1; y < (*height); ++y){
        for(x=threadIdx.x; x < (*width); x += blockDim.x){
            char left = ((x-1)>=0) ? cellData[(x-1) + (y-1) * (*width)] : 0;
            char middle = cellData[x + (y-1) * (*width)];
            char right = ((x+1) < *width) ? cellData[(x+1) + (y-1) * (*width)] : 0;

            //printf("left   : (%d, %d) => %d : %d\n", x-1, y-1, (x-1) + (y-1) * (*width), left);
            //printf("middle : (%d, %d) => %d : %d\n", x, y-1, (x) + (y-1) * (*width), middle);
            //printf("right  : (%d, %d) => %d : %d\n", x+1, y-1, (x+1) + (y-1) * (*width), right);
            applyRule(&left, &middle, &right, &cellData[x + y * (*width)]);
            //printf("res    : (%d, %d) => %d : %d\n", x, y, x + y * (*width), cellData[x+y*(*width)]);

            //printf("-----------\n");
        }
        __syncthreads();
    }

    /*
       for(y=0; y < *height; ++y){
       for(x=0; x < *width; ++x){
       printf("(%d, %d) = %d\n", x, y, cellData[x+y*(*width)]);
       }
       }
     */
}

void resetGame(Game* ptr, int type)
{
    const unsigned int SIZE = ptr->m_width / (CELL_SIZE);
    if(type == 0){
        unsigned int i = 0;
        for(i=0; i < SIZE; ++i){
            ptr->m_cellData_host[i] = rand() % 2;
        }
    }else if(type == 1){
        ptr->m_cellData_host[SIZE-1] = 1;
    }
    hipMemcpy(ptr->m_cellData_device, ptr->m_cellData_host, sizeof(char) * SIZE, hipMemcpyHostToDevice);
    computeCell<<<1, 256>>>(ptr->m_cellData_device, ptr->m_width_device, ptr->m_height_device);
    hipError_t err = hipGetLastError();
    if(err != hipSuccess){
        printf("Error : %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
    hipMemcpy(ptr->m_cellData_host, ptr->m_cellData_device, ptr->m_cellDataSize, hipMemcpyDeviceToHost);
}

void deleteGame(Game* ptr)
{
    SDL_DestroyWindow(ptr->m_window);
    SDL_DestroyRenderer(ptr->m_renderer);
    SDL_Quit();
    free(ptr->m_cellData_host);
    hipFree(ptr->m_cellData_device);
    hipFree(ptr->m_width_device);
    hipFree(ptr->m_height_device);
    free(ptr);
    ptr = NULL; 
}
